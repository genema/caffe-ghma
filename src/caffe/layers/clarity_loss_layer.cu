#include "hip/hip_runtime.h"
/*
* @Author: gehuama
* @Date:   2017-12-09 18:35:17
* @Last Modified by:   gehuama
* @Last Modified time: 2017-12-09 18:35:17
*/

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layers/loss_layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "math.h"

namespace caffe {

template <typename Dtype>
void ClarityLossLayer<Dtype>::Reshape(
  const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  LossLayer<Dtype>::Reshape(bottom, top);
  CHECK_EQ(bottom[0]->count(1), bottom[1]->count(1))
      << " CLARITY LOSS --> Inputs bottom[0] [1] must have the same dim.";
  CHECK_EQ(bottom[0]->count(1), bottom[2]->count(1))
      << " CLARITY LOSS --> Inputs bottom[0] [2] must have the same dim.";
  diff_.ReshapeLike(*bottom[0]);
  x_.ReshapeLike(*bottom[0]);
  x2_.ReshapeLike(*bottom[0]);
}

template <typename Dtype>
void ClarityLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  //caffe_copy(count, bottom[2]->gpu_data(), x_.mutable_gpu_data()); // num is the N in each batch
  //caffe_gpu_sqrt(count, x_.gpu_data(), x_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  //caffe_gpu_add_scalar(count, Dtype(0.000001), x_.mutable_gpu_data());
  Dtype loss = sqrt(dot^2+0.01^2) / bottom[0]->num(); 
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void ClarityLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    //printf(" propagate_down=%d, i=%d",propagate_down[i], i);
    if (propagate_down[i]) {
        const Dtype sign = (i == 0) ? 1 : -1;
        const Dtype alpha = 1 / (sign * top[0]->cpu_diff()[0]) / bottom[i]->num(); //(const Dtype*)diff_->cpu_data();
        caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                           // alpha
          diff_.gpu_data(),           // x
          Dtype(0),                        // beta
          bottom[i]->mutable_gpu_diff());  // y
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ClarityLossLayer);

}  // namespace caffe
